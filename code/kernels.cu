#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorNorm(double *a, double *result, int size) //Can be used for calculating the frobenius norm as well
{
    __shared__ double cache[1024];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int cacheIndex = threadIdx.x;
    double temp = 0.0;

    while(tid < size)
    {
        temp += a[tid] * a[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (cacheIndex < i)
        {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
    {
        atomicAdd(result, cache[0]);
    }
}

__global__ void matrixMultiply(double *A, double *B, double *C, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        double sum = 0.0;
        for (int i = 0; i < k; i++) {
            sum += A[row * k + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

__global__ void transposeMatrix(double* input, double* output, int rows, int cols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < rows && j < cols) {
        output[j * rows + i] = input[i * cols + j];
    }
}

__global__ void applyHouseholderToColumns(double* A, double* u, int m, int n, int k, int ldA) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= k && j < n) {
        // Calculate dot product u^T * A(:,j)
        double dot_product = 0.0;
        for (int i = k; i < m; ++i) {
            dot_product += u[i - k] * A[i * ldA + j];
        }

        // Apply transformation A = (I - 2uu^T) * A
        for (int i = k; i < m; ++i) {
            A[i * ldA + j] -= 2 * u[i - k] * dot_product;
        }
    }
}

__global__ void updateUMatrix(double* U, double* u, int m, int k, int ldU) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < m) {
        // Calculate dot product U(i,:) * u
        double dot_product = 0.0;
        for (int j = k; j < m; ++j) {
            dot_product += U[i * ldU + j] * u[j - k];
        }

        // Update U row
        for (int j = k; j < m; ++j) {
            U[i * ldU + j] -= 2 * dot_product * u[j - k];
        }
    }
}


__global__ void applyHouseholderToRows(double* A, double* u, int m, int n, int k, int ldA) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < m) {
        // Calculate dot product A(i,:) * u
        double dot_product = 0.0;
        for (int j = k + 1; j < n; ++j) {
            dot_product += A[i * ldA + j] * u[j - (k + 1)];
        }

        // Apply transformation A = A * (I - 2uu^T)
        for (int j = k + 1; j < n; ++j) {
            A[i * ldA + j] -= 2 * dot_product * u[j - (k + 1)];
        }
    }
}

__global__ void updateVMatrix(double* V, double* u, int n, int k, int ldV) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        // Calculate dot product V(i,:) * u
        double dot_product = 0.0;
        for (int j = k + 1; j < n; ++j) {
            dot_product += V[i * ldV + j] * u[j - (k + 1)];
        }

        // Update V row
        for (int j = k + 1; j < n; ++j) {
            V[i * ldV + j] -= 2 * dot_product * u[j - (k + 1)];
        }
    }
}
